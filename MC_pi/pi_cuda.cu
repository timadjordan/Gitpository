#include <stdio.h>
#include <cmath>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <ctime>
#include <chrono>

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}


__global__ void MCpoints(int* d_out, int n, int n_for, hiprandState* state)
{
    const int myID = blockIdx.x*blockDim.x + threadIdx.x; 

    hiprand_init(1234, myID, 0, &state[myID]);
    hiprandState localState = state[myID];

    d_out[myID] = 0;
    
    for (int i=0; i<n_for; i++)
    {
	float x = hiprand_uniform(&localState);
    	float y = hiprand_uniform(&localState);

    	if (sqrt(x*x+y*y)<1)
    	{
	  d_out[myID]++;
    	}
    	//printf("d_out[%d] = %d\n",myID,d_out[myID]);
    }
}

int main()
{

  const long int N = pow(2,31);
  const int B	   = 1024;
  const int TPB    = 32;
  const int T	   = (B*TPB);
  const int n_for  = N/T; 
  int counter = 0;

  int* out = (int*) calloc(T,sizeof(int));

  int* d_out = 0;
  hipMalloc(&d_out, T*sizeof(int));

  hiprandState *devStates;
  hipMalloc(&devStates, T*sizeof(int));

  auto t1 = std::chrono::high_resolution_clock::now();
  MCpoints<<<B, TPB>>>(d_out, T, n_for, devStates);
  gpuErrchk( hipMemcpy(out, d_out, T*sizeof(int), hipMemcpyDeviceToHost) );
  for (int i=0; i<T; i++)
  {
    counter += out[i];
  }
  double pi = 4*double(counter)/double(N);
  auto t2 = std::chrono::high_resolution_clock::now();
  auto t_elapsed = std::chrono::duration<double>(t2-t1).count();

  printf("samples = %ld\ncounter = %d\npi = %f\ntime elapsed: %1.4f sec\n",N,counter,pi,t_elapsed);

  hipFree(d_out);
  hipFree(devStates);
  free(out);

}
