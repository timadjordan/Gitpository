#include <stdio.h>
#include <cmath>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}


__global__ void MCpoints(int* d_out, int n, int n_for, hiprandState* state)
{
    const int myID = blockIdx.x*blockDim.x + threadIdx.x; 

    hiprand_init(1234, myID, 0, &state[myID]);
    hiprandState localState = state[myID];

    d_out[myID] = 0;
    
    for (int i=0; i<n_for; i++)
    {
	float x = hiprand_uniform(&localState);
    	float y = hiprand_uniform(&localState);

    	if (sqrt(x*x+y*y)<1)
    	{
	  d_out[myID]++;
    	}
    	//printf("d_out[%d] = %d\n",myID,d_out[myID]);
    }
}

int main()
{

  const long int N = pow(2,30);
  const int B	   = 1024;
  const int TPB    = 32;
  const int T	   = (B*TPB);
  const int n_for  = N/T; 
  int counter = 0;

  int* out = (int*) calloc(T,sizeof(int));

  int* d_out = 0;
  hipMalloc(&d_out, T*sizeof(int));

  hiprandState *devStates;
  hipMalloc(&devStates, T*sizeof(int));

  MCpoints<<<B, TPB>>>(d_out, T, n_for, devStates);
  gpuErrchk( hipMemcpy(out, d_out, T*sizeof(int), hipMemcpyDeviceToHost) );
  for (int i=0; i<T; i++)
  {
    counter += out[i];
  }

  double pi = 4*double(counter)/double(N);
  printf("samples = %ld\ncounter = %d\npi = %f\n",N,counter,pi);

  hipFree(d_out);
  hipFree(devStates);
  free(out);

}
