#include "hip/hip_runtime.h"
#include <stdio.h>
#include <cmath>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}


__global__ void MCpoints(int* d_out, int n, int n_for, hiprandState* state)
{
    const int myID = blockIdx.x*blockDim.x + threadIdx.x; 

    hiprand_init(1234, myID, 0, &state[myID]);
    hiprandState localState = state[myID];

    d_out[myID] = 0;
    
    for (i=0; i<n_for; i++)
    {
	float x = hiprand_uniform(&localState);
    	float y = hiprand_uniform(&localState);

    	if (sqrt(x*x+y*y)<1)
    	{
	  d_out[myID] += 1;
    	}
    	//printf("d_out[%d] = %d\n",myID,d_out[myID]);
    }
}

int main()
{

  const int N   = pow(2,16);
  const int TPB = 32; 
  int counter = 0;

  int* out = (int*) calloc(N,sizeof(int));

  int* d_out = 0;
  hipMalloc(&d_out, N*sizeof(int));

  hiprandState *devStates;
  hipMalloc(&devStates, N*sizeof(int));

  gpuErrchk( hipMemcpy(d_out, out, N*sizeof(int), hipMemcpyHostToDevice) );
  MCpoints<<<N/TPB, TPB>>>(d_out, N, devStates);
  printf("size = %lu\n",N*sizeof(int));
  gpuErrchk( hipMemcpy(out, d_out, N*sizeof(int), hipMemcpyDeviceToHost) );
  for (int i=0; i<N; i++)
  {
    counter += out[i];
  }

  double pi = 4*double(counter)/double(N);
  printf("samples = %d\ncounter = %d\npi = %f\n",N,counter,pi);

  hipFree(d_out);
  hipFree(devStates);
  free(out);

}
